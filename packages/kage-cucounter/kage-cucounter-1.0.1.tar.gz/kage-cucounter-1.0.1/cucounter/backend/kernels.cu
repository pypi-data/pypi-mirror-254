#include "hip/hip_runtime.h"
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "common.h"
#include "kernels.h"

namespace kernels {

namespace cg = cooperative_groups;

__device__ __forceinline__ static uint64_t word_reverse_complement(
    const uint64_t kmer, uint8_t kmer_size) 
{
  uint64_t res = ~kmer;
  res = ((res >> 2 & 0x3333333333333333) | (res & 0x3333333333333333) << 2);
  res = ((res >> 4 & 0x0F0F0F0F0F0F0F0F) | (res & 0x0F0F0F0F0F0F0F0F) << 4);
  res = ((res >> 8 & 0x00FF00FF00FF00FF) | (res & 0x00FF00FF00FF00FF) << 8);
  res = ((res >> 16 & 0x0000FFFF0000FFFF) | (res & 0x0000FFFF0000FFFF) << 16);
  res = ((res >> 32 & 0x00000000FFFFFFFF) | (res & 0x00000000FFFFFFFF) << 32);
  return (res >> (2 * (32 - kmer_size)));
}

__device__ __forceinline__ static uint64_t murmur_hash(uint64_t kmer) 
{
#ifdef __USE_MURMUR_HASH__
  kmer ^= kmer >> 33;
  kmer *= 0xff51afd7ed558ccd;
  kmer ^= kmer >> 33;
  kmer *= 0xc4ceb9fe1a85ec53;
  kmer ^= kmer >> 33;
#endif
  return kmer;
}

// ----- INITIALIZE -----

__global__ void initialize_hashtable_kernel(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity) 
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id >= size) 
  {
    return;
  }

  uint64_t insert_key = keys[thread_id];
  uint64_t hash = murmur_hash(insert_key) % capacity;

  while (true) 
  {
    unsigned long long int *table_key_ptr = 
      reinterpret_cast<unsigned long long int *>(&table.keys[hash]);
    uint64_t old = atomicCAS(table_key_ptr, kEmpty, insert_key);

    const bool inserted = (old == kEmpty || old == insert_key);

    if (inserted)
    {
      return;
    }
    hash = (hash + 1) % capacity;
  }
}

void initialize_hashtable(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity) 
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      initialize_hashtable_kernel, 0, 0));

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  initialize_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, size, capacity);
}

__global__ void cg_initialize_hashtable_kernel(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity) 
{
  int key_index = (blockIdx.x * blockDim.x + threadIdx.x) / cg_size;
  if (key_index >= size) 
  {
    return;
  }

  cg::thread_block_tile<cg_size> group = cg::tiled_partition<cg_size>(cg::this_thread_block());
  uint64_t insert_key = keys[key_index];
  uint64_t hash = murmur_hash(insert_key) % capacity;
  hash = (hash + group.thread_rank()) % capacity;

  while (true) 
  {
    uint64_t table_key = table.keys[hash];

    bool empty = (table_key == kEmpty);
    auto empty_mask = group.ballot(empty);
    while (empty_mask)
    {
      bool inserted = false;

      // Determine leader
      const int leader = __ffs(empty_mask) - 1;
      if (group.thread_rank() == leader)
      {
        unsigned long long int *table_key_ptr = 
          reinterpret_cast<unsigned long long int *>(&table.keys[hash]);
        const uint64_t old = atomicCAS(table_key_ptr, kEmpty, insert_key);

        inserted = (old == kEmpty || old == insert_key);
      }

      if (group.any(inserted))
      {
        return;
      }

      empty_mask ^= (1UL << leader);
    }

    hash = (hash + cg_size) % capacity;
  }
}

void cg_initialize_hashtable(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity) 
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      cg_initialize_hashtable_kernel, 0, 0));

  int grid_size = (size*cg_size) / thread_block_size + ((size*cg_size) % thread_block_size > 0);
  cg_initialize_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, size, capacity);
}

// ----- LOOKUP -----

__global__ void lookup_hashtable_kernel(const Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity) 
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id >= size)
  {
    return;
  }

  uint64_t lookup_key = keys[thread_id];
  uint64_t hash = murmur_hash(lookup_key) % capacity;

  while (true) 
  {
    uint64_t table_key = table.keys[hash];
    if (table_key == lookup_key || table_key == kEmpty) 
    {
      counts[thread_id] = (table_key == lookup_key) ? table.values[hash] : 0;
      return;
    }
    hash = (hash + 1) % capacity;
  }
}

void lookup_hashtable(const Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity) 
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      lookup_hashtable_kernel, 0, 0));

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  lookup_hashtable_kernel<<<grid_size, thread_block_size>>>(table, keys, counts, size, capacity);
}

__global__ void cg_lookup_hashtable_kernel(const Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity)
{
  int key_index = (blockIdx.x * blockDim.x + threadIdx.x) / cg_size;
  if (key_index >= size) 
  {
    return;
  }

  cg::thread_block_tile<cg_size> group = cg::tiled_partition<cg_size>(cg::this_thread_block());
  uint64_t lookup_key = keys[key_index];
  uint64_t hash = murmur_hash(lookup_key) % capacity;
  hash = (hash + group.thread_rank()) % capacity;

  while (true) 
  {
    uint64_t table_key = table.keys[hash];

    const bool hit = (lookup_key == table_key);
    const auto hit_mask = group.ballot(hit);
    if (hit_mask)
    {
      const int leader = __ffs(hit_mask) - 1;
      if (group.thread_rank() == leader)
      {
        counts[key_index] = table.values[hash];
      }
      return;
    }

    const bool empty = (table_key == kEmpty);
    const auto empty_mask = group.ballot(empty);
    if (empty_mask) {
      return;
    }

    hash = (hash + cg_size) % capacity;
  }
}

void cg_lookup_hashtable(const Table table, 
    const uint64_t *keys, uint32_t *counts, const uint32_t size, const uint32_t capacity)
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      cg_lookup_hashtable_kernel, 0, 0));

  int grid_size = (size*cg_size) / thread_block_size + ((size*cg_size) % thread_block_size > 0);
  lookup_hashtable_kernel<<<grid_size, thread_block_size>>>(
      table, keys, counts, size, capacity);
}

// ----- COUNT -----

__global__ void count_hashtable_kernel(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity,
    const bool count_revcomps, const uint8_t kmer_size) 
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id >= size)
  {
    return;
  }

  // Search for original key
  uint64_t insert_key = keys[thread_id];
  uint64_t hash = murmur_hash(insert_key) % capacity;

  while (true)
  {
    uint64_t table_key = table.keys[hash];
    if (table_key == kEmpty) 
    { 
      break; 
    }
    if (table_key == insert_key) 
    {
      atomicAdd((unsigned int *)&(table.values[hash]), 1);
      break;
    }
    hash = (hash + 1) % capacity;
  }

  if (count_revcomps)
  {
    // Search for reverse complement of key
    insert_key = word_reverse_complement(insert_key, kmer_size);
    hash = murmur_hash(insert_key) % capacity;

    while (true) 
    {
      uint64_t table_key = table.keys[hash];
      if (table_key == kEmpty) 
      { 
        return;
      }
      if (table_key == insert_key) 
      {
        atomicAdd((unsigned int *)&(table.values[hash]), 1);
        return;
      }
      hash = (hash + 1) % capacity;
    }
  }
}

void count_hashtable(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity,
    const bool count_revcomps, const uint8_t kmer_size) 
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      count_hashtable_kernel, 0, 0));

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  count_hashtable_kernel<<<grid_size, thread_block_size>>>(
      table, keys, size, capacity, count_revcomps, kmer_size);
}

__global__ void cg_count_hashtable_kernel(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity,
    const bool count_revcomps, const uint8_t kmer_size) 
{
  int key_index = (blockIdx.x * blockDim.x + threadIdx.x) / cg_size;
  if (key_index >= size) 
  {
    return;
  }

  cg::thread_block_tile<cg_size> group = cg::tiled_partition<cg_size>(cg::this_thread_block());
  uint64_t insert_key = keys[key_index];
  uint64_t hash = murmur_hash(insert_key) % capacity;
  hash = (hash + group.thread_rank()) % capacity;

  while (true) 
  {
    uint64_t table_key = table.keys[hash];

    const bool hit = (insert_key == table_key);
    const auto hit_mask = group.ballot(hit);
    if (hit_mask)
    {
      const int leader = __ffs(hit_mask) - 1;
      if (group.thread_rank() == leader)
      {
        atomicAdd((unsigned int *)&(table.values[hash]), 1);
      }
      return;
    }

    const bool empty = (table_key == kEmpty);
    const auto empty_mask = group.ballot(empty);
    if (empty_mask) {
      return;
    }

    hash = (hash + cg_size) % capacity;
  }
}

void cg_count_hashtable(Table table, 
    const uint64_t *keys, const uint32_t size, const uint32_t capacity,
    const bool count_revcomps, const uint8_t kmer_size) 
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      cg_count_hashtable_kernel, 0, 0));

  int grid_size = (size*cg_size) / thread_block_size + ((size*cg_size) % thread_block_size > 0);
  count_hashtable_kernel<<<grid_size, thread_block_size>>>(
      table, keys, size, capacity, count_revcomps, kmer_size);
}

// ----- PROBE LENGHT -----

__global__ void get_probe_lengths_kernel(const Table table, 
    const uint64_t *keys, uint32_t *lengths, 
    const uint32_t size, const uint32_t capacity) 
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id >= size)
  {
    return;
  }

  uint64_t key = keys[thread_id];
  uint64_t hash = murmur_hash(key) % capacity;
  uint32_t probes = 1;

  while (true) 
  {
    uint64_t table_key = table.keys[hash];
    if (table_key == key || table_key == kEmpty) 
    {
      lengths[thread_id] = probes;
      return;
    }
    hash = (hash + 1) % capacity;
    probes++;
  }
}

void get_probe_lengths(const Table table, 
    const uint64_t *keys, uint32_t *lengths, 
    const uint32_t size, const uint32_t capacity)
{
  int min_grid_size;
  int thread_block_size;
  cuda_errchk(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, 
      get_probe_lengths_kernel, 0, 0));

  int grid_size = size / thread_block_size + (size % thread_block_size > 0);
  get_probe_lengths_kernel<<<grid_size, thread_block_size>>>(
      table, keys, lengths, size, capacity);
}

} // kernels
