#include <iostream>
#include <sstream>
#include <inttypes.h>
#include <string>

#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "hashtable.h"

HashTable::HashTable(const uint64_t *keys, const bool cuda_keys, 
    const uint32_t size, const uint32_t capacity) 
{
  initialize(keys, cuda_keys, size, capacity);
}

void HashTable::initialize(const uint64_t *keys, const bool cuda_keys, 
    const uint32_t size, const uint32_t capacity) 
{
  capacity_m = capacity;
  size_m = size;

  // Allocate the table
  cuda_errchk(hipMalloc(&table_m.keys, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMemset(table_m.keys, 0xff, sizeof(uint64_t)*capacity));
  cuda_errchk(hipMalloc(&table_m.values, sizeof(uint32_t)*capacity));
  cuda_errchk(hipMemset(table_m.values, 0, sizeof(uint32_t)*capacity));

  uint64_t *d_keys;
  if (!cuda_keys) 
  {
    cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
    cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

#ifdef _USE_COOPERATIVE_GROUPS
  kernels::cg_initialize_hashtable(table_m, cuda_keys ? keys : d_keys, size, capacity);
#else
  kernels::initialize_hashtable(table_m, cuda_keys ? keys : d_keys, size, capacity);
#endif

  if (!cuda_keys) 
  {
    cuda_errchk(hipFree(d_keys));
  }
}

void HashTable::get(const uint64_t *keys, uint32_t *counts, uint32_t size) const 
{
  uint64_t *d_keys;
  uint32_t *d_counts;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMalloc(&d_counts, sizeof(uint32_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

#ifdef _USE_COOPERATIVE_GROUPS
  kernels::cg_lookup_hashtable(table_m, d_keys, d_counts, size, capacity_m); 
#else
  kernels::lookup_hashtable(table_m, d_keys, d_counts, size, capacity_m); 
#endif

  cuda_errchk(hipMemcpy(counts, d_counts, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));
  cuda_errchk(hipFree(d_keys));
  cuda_errchk(hipFree(d_counts));
}

void HashTable::cu_get(const uint64_t *keys, uint32_t *counts, uint32_t size) const 
{
#ifdef _USE_COOPERATIVE_GROUPS
  kernels::cg_lookup_hashtable(table_m, keys, counts, size, capacity_m); 
#else
  kernels::lookup_hashtable(table_m, keys, counts, size, capacity_m); 
#endif
}

void HashTable::count(const uint64_t *keys, const uint32_t size,
    const bool count_revcomps, const uint8_t kmer_size) 
{
  uint64_t *d_keys;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

#ifdef _USE_COOPERATIVE_GROUPS
  kernels::cg_count_hashtable(table_m, d_keys, size, capacity_m, count_revcomps, kmer_size);
#else
  kernels::count_hashtable(table_m, d_keys, size, capacity_m, count_revcomps, kmer_size);
#endif

  cuda_errchk(hipFree(d_keys));
}

void HashTable::cu_count(const uint64_t *keys, const uint32_t size,
    const bool count_revcomps, const uint8_t kmer_size) 
{
#ifdef _USE_COOPERATIVE_GROUPS
  kernels::cg_count_hashtable(table_m, keys, size, capacity_m, count_revcomps, kmer_size);
#else
  kernels::count_hashtable(table_m, keys, size, capacity_m, count_revcomps, kmer_size);
#endif
}

void HashTable::get_probe_lengths(
    const uint64_t *keys, uint32_t *lengths, const uint32_t size) const 
{
  uint64_t *d_keys;
  uint32_t *d_lengths;
  cuda_errchk(hipMalloc(&d_keys, sizeof(uint64_t)*size));
  cuda_errchk(hipMalloc(&d_lengths, sizeof(uint32_t)*size));
  cuda_errchk(hipMemcpy(d_keys, keys, sizeof(uint64_t)*size, hipMemcpyHostToDevice));

  kernels::get_probe_lengths(table_m, d_keys, d_lengths, size, capacity_m);
  cuda_errchk(hipMemcpy(lengths, d_lengths, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));

  cuda_errchk(hipFree(d_keys));
  cuda_errchk(hipFree(d_lengths));
}

void HashTable::cu_get_probe_lengths(
    const uint64_t *keys, uint32_t *lengths, const uint32_t size) const
{
  kernels::get_probe_lengths(table_m, keys, lengths, size, capacity_m);
}

std::string HashTable::to_string(const bool full) const 
{
  int print_size = (capacity_m < 40) ? capacity_m : 40;

  uint64_t *keys = new uint64_t[capacity_m];
  uint32_t *values = new uint32_t[capacity_m];
  cuda_errchk(hipMemcpy(
        keys, table_m.keys, sizeof(uint64_t)*capacity_m, hipMemcpyDeviceToHost));
  cuda_errchk(hipMemcpy(
        values, table_m.values, sizeof(uint32_t)*capacity_m, hipMemcpyDeviceToHost));

  std::ostringstream oss;
  std::ostringstream keys_oss;
  std::ostringstream values_oss;

  keys_oss << "[";
  values_oss << "[";
  uint32_t elements = 0;
  for (int i = 0; i < capacity_m; i++) 
  {
    uint64_t key = keys[i];
    uint32_t value = values[i];

    if (key == kEmpty && !full) { continue; }

    if (elements != 0) 
    { 
      keys_oss << ", "; 
      values_oss << ", "; 
    }

    if (key == kEmpty)
    {
      keys_oss << "kEmp";
    }
    else
    {
      keys_oss << key;
    }

    //keys_oss << ((key == kEmpty) ? "E" : key);
    values_oss << value;
    
    elements++;
    if (elements >= print_size && !full) { break; }
  }
  keys_oss << "]";
  values_oss << "]";

  oss << "Counter(" << keys_oss.str() << ", " << values_oss.str();
  oss << ", size=" << size_m << ", capacity=" << capacity_m << ")";

  delete[] keys;
  delete[] values;

  return oss.str();
}
